#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

bool debug = false;

template <typename Dtype>
__device__ void swap_array_values(Dtype * array, int i, int j) {
  Dtype tmp_value = array[i];
  array[i] = array[j];
  array[j] = tmp_value;
}

template <typename Dtype, typename T>
__device__ void insertion_sort_ascend(Dtype array[], T follow_array[], const int length) {
  for (int array_index = 1 ; array_index < length; array_index++) {
    int sortid = array_index;
    while (sortid > 0 && array[sortid] < array[sortid - 1]) {
      swap_array_values(array, sortid, sortid - 1);
      swap_array_values(follow_array, sortid, sortid - 1);
      --sortid;
    }
  }
}

template <typename Dtype, typename T>
__device__ void insertion_sort_descend(Dtype array[], T follow_array[], const int length) {
  for (int array_index = 1 ; array_index < length; array_index++) {
    int sortid = array_index;
    while (sortid > 0 && array[sortid] > array[sortid - 1]) {
      swap_array_values(array, sortid, sortid - 1);
      swap_array_values(follow_array, sortid, sortid - 1);
      --sortid;
    }
  }
}

template <typename Dtype, typename T>
__device__ void heap_adjust_ascend(Dtype array[], T follow_array[], int i, const int length) {
  while (2 * i + 1 < length) {
    int child_index = 2 * i + 1;
    if (child_index < length - 1 && array[child_index + 1] > array[child_index])
      ++child_index;
    if (array[i] < array[child_index]) {
      swap_array_values(array, i, child_index);
      swap_array_values(follow_array, i, child_index);
    } else
      break;
    i = child_index;
  }
}

template <typename Dtype, typename T>
__device__ void heap_sort_ascend(Dtype array[], T follow_array[], const int length) {
  if (length == 1)
    return;
  for (int i = (length - 2) / 2; i >= 0; --i)
    heap_adjust_ascend(array, follow_array, i, length);
  for (int i = length - 1; i > 0; --i) {
    swap_array_values(array, i, 0);
    swap_array_values(follow_array, i, 0);
    heap_adjust_ascend(array, follow_array, 0, i);
  }
}

template <typename Dtype, typename T>
__device__ void heap_adjust_descend(Dtype array[], T follow_array[], int i, const int length) {
  while (2 * i + 1 < length) {
    int child_index = 2 * i + 1;
    if (child_index < length - 1 && array[child_index + 1] < array[child_index])
      ++child_index;
    if (array[i] > array[child_index]) {
      swap_array_values(array, i, child_index);
      swap_array_values(follow_array, i, child_index);
    } else
      break;
    i = child_index;
  }
}

template <typename Dtype, typename T>
__device__ void heap_sort_descend(Dtype array[], T follow_array[], const int length) {
  if (length == 1)
    return;
  for (int i = (length - 2) / 2; i >= 0; --i)
    heap_adjust_descend(array, follow_array, i, length);
  for (int i = length - 1; i > 0; --i) {
    swap_array_values(array, i, 0);
    swap_array_values(follow_array, i, 0);
    heap_adjust_descend(array, follow_array, 0, i);
  }
}

namespace caffe {

template <typename Dtype>
void print_blob_data(const Blob<Dtype> * blob, const Dtype * blob_data) {
  int data_index = 0;
  for (int n = 0; n < blob->num(); ++n) {
    for (int c = 0; c < blob->channels(); ++c) {
      std::cout << "num:" << n << ", channel:" << c << std::endl;
      for (int h = 0; h < blob->height(); ++h) {
        for (int w = 0; w < blob->width(); ++w) {
          std::cout << blob_data[data_index++];
          if (blob->width() != w + 1)
            std::cout << " ";
          else
            std::cout << std::endl;
        }
      }
    }
  }
}

template <typename Dtype>
__global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w, Dtype* top_data,
    int* mask, Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_data[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_data[maxidx];
        }
      }
    }
    top_data[index] = maxval;
    if (mask) {
      mask[index] = maxidx;
    } else {
      top_mask[index] = maxidx;
    }
  }
}

template <typename Dtype>
__global__ void KMaxPoolForward(const int n_pooling, const Dtype* bottom_data,
                                const int bottom_num, const int bottom_channels,
                                const int bottom_height, const int bottom_width,
                                const int pooled_height, const int pooled_width,
                                const int top_height, const int top_width,
                                const int kernel_h, const int kernel_w,
                                const int stride_h, const int stride_w,
                                const int pad_h, const int pad_w,
                                const int pool_direction, const int top_k,
                                Dtype* top_data, int* mask, Dtype* top_mask,
                                Dtype* pool_data, int* pool_ids,
                                const int sort_method, const int max_strategy) {
  CUDA_KERNEL_LOOP(index, n_pooling) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % bottom_channels;
    int n = index / pooled_width / pooled_height / bottom_channels;

    /*  pooling region on bottom data  */
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, bottom_height);
    int wend = min(wstart + kernel_w, bottom_width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    int length = (hend - hstart) * (wend - wstart);
    bottom_data += (n * bottom_channels + c) * bottom_height * bottom_width;
    pool_data += index * kernel_h * kernel_w;
    pool_ids += index * kernel_h * kernel_w;

    int array_index = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int position_offset = h * bottom_width + w;
        if (max_strategy == PoolingParameter_MaxStrategy_ORIGINAL) {
          pool_data[array_index] = bottom_data[position_offset];
        } else if (max_strategy == PoolingParameter_MaxStrategy_ABSOLUTE) {
          if (bottom_data[position_offset] >= 0)
            pool_data[array_index] = bottom_data[position_offset];
          else
            pool_data[array_index] = -bottom_data[position_offset];
        } else {
          //TODO echo error and exit
        }
        pool_ids[array_index] = position_offset;
        ++array_index;
      }
    }

    /*  sort k max  */
    if (sort_method == PoolingParameter_SortMethod_HEAP) {
      heap_sort_descend(pool_data, pool_ids, length);
      heap_sort_ascend(pool_ids, pool_data, top_k);
    } else if (sort_method == PoolingParameter_SortMethod_INSERTION) {
      insertion_sort_descend(pool_data, pool_ids, length);
      insertion_sort_ascend(pool_ids, pool_data, top_k);
    }

    /*  put k max onto top data  */
    for (int array_index = 0; array_index < top_k; ++array_index) {
      int top_data_index = -1;
      if (pool_direction == PoolingParameter_PoolDirection_HORIZONTAL)
        top_data_index = (n * bottom_channels + c) * top_height * top_width +
          ph * top_width + pw * top_k + array_index;
      else if (pool_direction == PoolingParameter_PoolDirection_VERTICAL)
        top_data_index = (n * bottom_channels + c) * top_height * top_width +
          (ph * top_k + array_index) * top_width + pw;

      top_data[top_data_index] = pool_data[array_index];

      if (mask)
        mask[top_data_index] = pool_ids[array_index];
      else
        top_mask[top_data_index] = pool_ids[array_index];
    }
  }
}

template <typename Dtype>
__global__ void AvePoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_data[h * width + w];
      }
    }
    top_data[index] = aveval / pool_size;
  }
}

template <typename Dtype>
__global__ void StoPoolForwardTrain(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* rand_idx, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + kernel_w, width);
    Dtype cumsum = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
      }
    }
    float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_data[h * width + w];
          return;
        }
      }
    }
  }
}


template <typename Dtype>
__global__ void StoPoolForwardTest(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + kernel_w, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    Dtype cumsum = FLT_MIN;
    Dtype cumvalues = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        cumvalues += bottom_data[h * width + w] * bottom_data[h * width + w];
      }
    }
    top_data[index] = cumvalues / cumsum;
  }
}


template <typename Dtype>
void PoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  int count = (*top)[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top->size() > 1;
  int* mask = NULL;
  Dtype* top_mask = NULL;

  int single_pooled_height = -1;
  int single_pooled_width = -1;
  int n_pooling = -1;
  Dtype * pool_data;
  int * pool_ids;

  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = (*top)[1]->mutable_gpu_data();
    } else {
      mask = max_idx_.mutable_gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data,
        mask, top_mask);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AvePoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    if (Caffe::phase() == Caffe::TRAIN) {
      // We need to create the random index as well.
      caffe_gpu_rng_uniform(count, Dtype(0), Dtype(1),
                            rand_idx_.mutable_gpu_data());
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTrain<Dtype><<<CAFFE_GET_BLOCKS(count),
                                   CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_,
          rand_idx_.mutable_gpu_data(), top_data);
    } else {
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTest<Dtype><<<CAFFE_GET_BLOCKS(count),
                                  CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, top_data);
    }
    break;
  case PoolingParameter_PoolMethod_KMAX:
    if (use_top_mask) {
      top_mask = (*top)[1]->mutable_gpu_data();
    } else {
      mask = max_idx_.mutable_gpu_data();
    }

    if (this->layer_param_.pooling_param().pool_direction() == PoolingParameter_PoolDirection_HORIZONTAL) {
      single_pooled_height = pooled_height_;
      single_pooled_width = pooled_width_ / this->layer_param_.pooling_param().top_k();
    } else if (this->layer_param_.pooling_param().pool_direction() == PoolingParameter_PoolDirection_VERTICAL) {
      single_pooled_height = pooled_height_ / this->layer_param_.pooling_param().top_k();
      single_pooled_width = pooled_width_;
    }
    n_pooling = count / this->layer_param_.pooling_param().top_k();

    hipMalloc((void**)&pool_data, sizeof(Dtype) * n_pooling * kernel_h_ * kernel_w_);
    hipMalloc((void**)&pool_ids, sizeof(int) * n_pooling * kernel_h_ * kernel_w_);
    // NOLINT_NEXT_LINE(whitespace/operators)
    KMaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(n_pooling), CAFFE_CUDA_NUM_THREADS>>>(
      n_pooling, bottom_data, bottom[0]->num(), channels_,
      height_, width_, single_pooled_height, single_pooled_width, pooled_height_, pooled_width_,
      kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
      this->layer_param_.pooling_param().pool_direction(),
      this->layer_param_.pooling_param().top_k(),
      top_data, mask, top_mask, pool_data, pool_ids,
      this->layer_param_.pooling_param().sort_method(),
      this->layer_param_.pooling_param().max_strategy());
    hipFree(pool_data);
    hipFree(pool_ids);
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const Dtype* top_mask, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart =
        (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    int pwstart =
        (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    int offset = (n * channels + c) * pooled_height * pooled_width;
    top_diff += offset;
    if (mask) {
      mask += offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (mask[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff[ph * pooled_width + pw];
          }
        }
      }
    } else {
      top_mask += offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (top_mask[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff[ph * pooled_width + pw];
          }
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
__global__ void KMaxPoolBackward(const int bottom_count, const Dtype* top_diff,
                                 const int* mask, const Dtype* top_mask,
                                 const int num, const int channels,
                                 const int bottom_height, const int bottom_width,
                                 const int top_height, const int top_width,
                                 const int kernel_h, const int kernel_w,
                                 const int stride_h, const int stride_w,
                                 const int pad_h, const int pad_w,
                                 const int pool_direction, const int top_k,
                                 Dtype* bottom_diff) {
  int pooled_height = 0;
  int pooled_width = 0;
  if (pool_direction == PoolingParameter_PoolDirection_HORIZONTAL) {
    pooled_height = top_height;
    pooled_width = top_width / top_k;
  } else if (pool_direction == PoolingParameter_PoolDirection_VERTICAL) {
    pooled_height = top_height / top_k;
    pooled_width = top_width;
  }

  const int nthreads = bottom_count;
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % bottom_width;
    int h = (index / bottom_width) % bottom_height;
    int c = (index / bottom_width / bottom_height) % channels;
    int n = index / bottom_width / bottom_height / channels;
    int phstart = (h + pad_h < kernel_h) ?
      0 : (h + pad_h - kernel_h) / stride_h + 1;
    int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    int pwstart = (w + pad_w < kernel_w) ?
      0 : (w + pad_w - kernel_w) / stride_w + 1;
    int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    if (pool_direction == PoolingParameter_PoolDirection_HORIZONTAL) {
      pwstart *= top_k;
      pwend *= top_k;
    } else if (pool_direction == PoolingParameter_PoolDirection_VERTICAL) {
      phstart *= top_k;
      phend *= top_k;
    }

    Dtype gradient = 0;
    int offset = (n * channels + c) * top_height * top_width;
    top_diff += offset;
    if (mask) {
      mask += offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (mask[ph * top_width + pw] == h * bottom_width + w) {
            gradient += top_diff[ph * top_width + pw];
          }
        }
      }
    } else {
      top_mask += offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (top_mask[ph * top_width + pw] == h * bottom_width + w) {
            gradient += top_diff[ph * top_width + pw];
          }
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
__global__ void AvePoolBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width + pad_w;
    int h = (index / width) % height + pad_h;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int phend = min(h / stride_h + 1, pooled_height);
    int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, height + pad_h);
        int wend = min(wstart + kernel_w, width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff[ph * pooled_width + pw] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}


template <typename Dtype>
__global__ void StoPoolBackward(const int nthreads,
    const Dtype* rand_idx, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int phend = min(h / stride_h + 1, pooled_height);
    int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    rand_idx += (n * channels + c) * pooled_height * pooled_width;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        gradient += top_diff[ph * pooled_width + pw] *
            (index == static_cast<int>(rand_idx[ph * pooled_width + pw]));
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
void PoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  const int count = (*bottom)[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  const int* mask = NULL;
  const Dtype* top_mask = NULL;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->gpu_data();
    } else {
      mask = max_idx_.gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, top_mask, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_,
        kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        bottom_diff);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AvePoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    // NOLINT_NEXT_LINE(whitespace/operators)
    StoPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, rand_idx_.gpu_data(), top_diff,
        top[0]->num(), channels_, height_, width_, pooled_height_,
        pooled_width_, kernel_h_, kernel_w_, stride_h_, stride_w_,
        bottom_diff);
    break;
  case PoolingParameter_PoolMethod_KMAX:
    if (use_top_mask) {
      top_mask = top[1]->gpu_data();
    } else {
      mask = max_idx_.gpu_data();
    }
    if (debug) {
      std::cout << "backward phase" << std::endl;
      std::cout << "top data" << std::endl;
      print_blob_data(top[0], top[0]->cpu_data());
      std::cout << "top mask" << std::endl;
      print_blob_data(&max_idx_, max_idx_.cpu_data());
      std::cout << "top diff" << std::endl;
      print_blob_data(top[0], top[0]->cpu_diff());
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    KMaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, mask, top_mask, top[0]->num(), channels_,
      height_, width_, pooled_height_, pooled_width_,
      kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
      this->layer_param_.pooling_param().pool_direction(),
      this->layer_param_.pooling_param().top_k(),
      bottom_diff);
    if (debug) {
      std::cout << "bottom diff" << std::endl;
      print_blob_data((*bottom)[0], (*bottom)[0]->cpu_diff());
    }
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_CLASS(PoolingLayer);


}  // namespace caffe
